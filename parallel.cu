#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

void image_save(int *original, int rows, int cols, char* name)
{
	unsigned char * image_final = (unsigned char*)calloc(rows*cols, sizeof(char)) ;
	for(int i = 1 ; i < rows ; i++) {
		for (int j = 1 ; j < cols ; j++ ) {
			image_final[(i-1)*cols + j-1] = (unsigned char)original[(i-1)*cols + j-1] ;

		}
	}
	stbi_write_png(name, rows, cols, 1, (const void*)image_final, rows);
}
__global__ void get_global_variance(int *local_variance, int *local_mean, int *image, int *image_filter, int *variance, int *r1, int *c1)
{
  int cols = *c1;
  int var = *variance;
  int r = blockIdx.x;
  int c = threadIdx.x;

  if(local_variance[r * cols + c] < var)
    local_variance[r * cols + c] = var;
  image_filter[r * cols + c] = image[r * cols + c] - (var / local_variance[r * cols + c]) * (image[r * cols + c] - local_mean[r * cols + c]);
}
int get_sum2(int *arr, int rows, int cols)
{
  int temp_sum = 0;
  for(int i = 0; i < rows; i++)
    for(int j = 0; j < cols; j++)
      temp_sum += arr[i * cols + j];
  return temp_sum;
}
__global__ void square_matrix2(int *image, int *image_sq, int *rows, int *cols) // parallelized matrix square fn called for calculating global variance
{
  int row_id = blockIdx.x ;
  int col_id = threadIdx.x ;
  int columns = *cols ;
  int sum = 0 ;
  for(int k = 0; k < columns ; k++)
	  sum = sum + image[row_id*columns + k]*image[col_id*columns + k] ;
  image_sq[row_id *columns + col_id] = sum ;

}
__device__ void square_matrix1(int *mat,int *result ,int rows, int cols) // non parallelisable square function called in compute_local_mean_variance
{
  int temp_sum = 0 ;
  for(int i = 0; i < rows; i++) {
	  for(int j = 0; j < cols; j++) {
		  temp_sum = 0 ;
		  for(int k = 0; k < cols; k++)
			  temp_sum = temp_sum + mat[i*cols + k] * mat[j*cols + k] ;
		  result[i*cols + j] = temp_sum ;
	  }
  }
}
__device__ int get_sum(int *arr, int rows, int cols)
{
  int temp_sum = 0;
  for(int i = 0; i < rows; i++)
    for(int j = 0; j < cols; j++)
      temp_sum += arr[i * cols + j];
  return temp_sum;
}
__device__ int* get_neighbours(int *image, int curr_row, int curr_col, int cols)
{
  int *near = (int *)malloc(sizeof(int) * 9);
  int next = 0;
  for(int i = curr_row - 1; i < curr_row + 2; i++)
  {
    for(int j = curr_col - 1; j < curr_col + 2; j++)
    {
      near[next] = image[i * cols + j];
      next++;
    }
  }
  return near;
}

__global__ void compute_local_mean_variance(int *image_pad, int *local_mean, int *local_variance, int *rows, int *cols)
{
    int r = blockIdx.x;
    int c = threadIdx.x;
    int near_sq[9] ;
    if(r != 0 && c != 0)
    {
      int *near = get_neighbours(image_pad, r, c, (*cols + 1));
      int curr_mean = get_sum(near, 3, 3) / 9;
      local_mean[(r - 1) * (*cols) + (c - 1)] = curr_mean;
      square_matrix1(near, near_sq , 3, 3);
      local_variance[(r - 1) * (*cols) + (c - 1)] = get_sum(near_sq, 3, 3) / 9 - curr_mean;
    }
}
__global__ void image_padding(int *image, int *image_pad, int *rows, int *cols)
{
  int r = blockIdx.x;
  int c = threadIdx.x;

  if(r != 0 && c != 0 && r != *rows - 1 && c != *cols - 1)
    image_pad[r * (*cols) + c] = image[(r - 1) * (*cols - 1) + c - 1];
  else
    image_pad[r * (*cols) + c] = 0;
}
__global__ void loadIMG(char *temp_image, int *image, int *rows, int *cols)
{
  int r = blockIdx.x;
  int c = threadIdx.x;

  image[r * (*cols) + c] = (int) temp_image[r * (*cols) + c];
}
__global__ void sobel_horizontal(int *image_final, int *image_pad, int *sobel, int *r1, int *c1)
{
  int cols = *c1;

  int r = blockIdx.x;
  int c = threadIdx.x;
  int temp = 0;
  if(r > 0 && c > 0)
  {
    int *near = get_neighbours(image_pad, r, c, cols + 1);
    for(int k = 0; k < 9; k++)
      temp += near[k] * sobel[k];
    image_final[(r - 1) * cols + (c - 1)] = temp;
//    if ( temp > 0 )
//    	printf(" %d \n", temp) ;
  }

}
void err(int checker) {
	hipError_t errchck = hipGetLastError() ;
	if (errchck != hipSuccess )
		 printf(" %d  %s \n" , checker , hipGetErrorString(errchck ) ) ;
}
int main()
{
  int variance, rows, cols, bpp;
  char name[100] ;
  // 1) Imread
  unsigned char *temp_image = stbi_load("logo.png", &rows, &cols, &bpp, 1);
  int image[rows * cols];

    // Parallel conversion of char image to int image
  int *p_image;
  char *p_temp_image;
  int *p_row, *p_col;
  int checkers = 0 ;
  hipMalloc((void **)&p_image, sizeof(int) * rows * cols);

  hipMalloc((void **)&p_temp_image, sizeof(char) * rows * cols);
  hipMalloc((void **)&p_row, sizeof(int));
  hipMalloc((void **)&p_col, sizeof(int));

  hipMemcpy(p_temp_image, temp_image, sizeof(char) * rows * cols, hipMemcpyHostToDevice);

  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);


  loadIMG<<<rows, cols>>>(p_temp_image, p_image, p_row, p_col);

  hipMemcpy(image, p_image, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);


  // Declarations
  int *image_sq = (int *)malloc(sizeof(int) * rows * cols);
  int sobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
  int local_mean[rows * cols];
  int local_variance[rows * cols];
  int image_filter[rows * cols];

  // 2) Padding the Image
  int image_pad[(rows+1)*(cols+1)];
  int *p_image_pad;

  // Parallelizing image_padding function
  hipMalloc((void **)&p_image_pad, sizeof(int) * (rows + 1) * (cols + 1));


  hipMemcpy(p_image, image, sizeof(int) * rows * cols, hipMemcpyHostToDevice);
  rows += 1;
  cols += 1;
  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);

  image_padding<<<rows,cols>>>(p_image, p_image_pad, p_row, p_col);
  hipMemcpy(image_pad, p_image_pad, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);

  rows -= 1;
  cols -= 1;

  // 3) Computing Local Mean and Local Variance
  // Note : This is not completely parallelized due to data dependency

  int *p_local_mean, *p_local_variance;

  hipMalloc((void **)&p_local_mean, sizeof(int) * (rows) * (cols));
  hipMalloc((void **)&p_local_variance, sizeof(int) * (rows) * (cols));


  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(p_image_pad, image_pad, sizeof(int) * (rows + 1) * (cols + 1), hipMemcpyHostToDevice);

  compute_local_mean_variance<<<rows, cols>>>(p_image_pad, p_local_mean, p_local_variance, p_row, p_col);

  hipMemcpy(local_mean, p_local_mean, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);

  hipMemcpy(local_variance, p_local_variance, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);

  strcpy(name, "localmean.png");
	image_save(local_mean, rows, cols, name);
	strcpy(name, "localvar.png");
	image_save(local_variance, rows, cols, name);
  printf("Function 3\n\n");
  // 4) Get Global Variance
  int *p_image_sq;

  // Square Matrix2 Function
  hipMalloc((void **)&p_image_sq, sizeof(int) * rows * cols);

  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(p_image, image, sizeof(int) * rows * cols, hipMemcpyHostToDevice);


  square_matrix2<<<rows, cols>>>(p_image, p_image_sq, p_row, p_col);

  hipMemcpy(image_sq, p_image_sq, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);

  hipFree(p_image) ;

  hipFree(p_image_sq);



  // Get Sum2 Function doesn't need to be parallelized
  variance = get_sum2(image_sq , 3, 3) - get_sum2(image , 3 , 3) ;
  variance = variance / (rows * cols);
  int *p_image_filter, *p_variance;

  hipMalloc((void **)&p_image_filter, sizeof(int) * rows * cols);

  hipMalloc((void **)&p_variance, sizeof(int));


  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(p_variance, &variance, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(p_image, image, sizeof(int) * rows * cols, hipMemcpyHostToDevice);

  hipMemcpy(p_local_mean, local_mean, sizeof(int) * rows * cols, hipMemcpyHostToDevice);

  hipMemcpy(p_local_variance, local_variance, sizeof(int) * rows * cols, hipMemcpyHostToDevice);


  get_global_variance<<<rows, cols>>>(p_local_variance, p_local_mean, p_image, p_image_filter, p_variance, p_row, p_col);

  hipMemcpy(image_filter, p_image_filter, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);

  strcpy(name, "noise_removed.png");
  image_save(image_filter, rows, cols, name);
  // 5) Apply horizontal sobel filter for edge detection

  // Image padding


  hipMemcpy(p_image_filter, image_filter, sizeof(int) * rows * cols, hipMemcpyHostToDevice);

  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);

  rows += 1;
  cols += 1;
  image_padding<<<rows, cols>>>(p_image_filter, p_image_pad, p_row, p_col);

  hipMemcpy(image_pad, p_image_pad, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);
  strcpy(name, "padded_img.png");
  image_save(image_pad, rows, cols, name);
  rows -= 1;
  cols -= 1;

  printf("Function 5.1\n\n");




  hipFree(p_local_variance);
  hipFree(p_local_mean);
  hipFree(p_image);

  int image_final[rows*cols] ;
  int *p_image_final;
  int *p_sobel;

  hipMalloc((void **)&p_image_final, sizeof(int) * rows * cols);
  hipMalloc((void **)&p_sobel, sizeof(int) * 9);

  hipMemcpy(p_image_pad, image_pad, sizeof(int)*(rows+1)*(cols+1), hipMemcpyHostToDevice);
  hipMemcpy(p_row, &rows, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(p_col, &cols, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(p_sobel, sobel, sizeof(int) * 9, hipMemcpyHostToDevice);

  sobel_horizontal<<<rows, cols>>>(p_image_final, p_image_pad, p_sobel, p_row, p_col);

  hipMemcpy(image_final, p_image_final, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
  err(checkers++) ;

  printf("\n\nFunction 5.2 , %d \n\n" , checkers);
  strcpy(name, "final_image.png");
  image_save(image_final, rows, cols, name);
  printf(" Processing complete , open final_image.png to see results \n");

  hipFree(p_row);
  hipFree(p_col);
  hipFree(p_sobel);
  hipFree(p_image_pad);
  hipFree(p_image_final);
  hipFree(p_local_variance);
  hipFree(p_local_mean);
  hipFree(p_image);
	return 0 ;
}


